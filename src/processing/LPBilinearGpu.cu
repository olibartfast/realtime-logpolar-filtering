#include "hip/hip_runtime.h"
#include "rtlp/processing/LPBilinearGpu.hpp"
#include "../kernels/LPBilinearGpuKernel.cu"

namespace rtlp {
namespace processing {

LPBilinearGpu::LPBilinearGpu(rtlp::core::Image *i, bool inv):LogPolar(i,inv){}

LPBilinearGpu::~LPBilinearGpu()
{
 hipFree(xc_d);
 hipFree(yc_d);
 hipFree(e_d);
 hipFree(n_d);	

}

void LPBilinearGpu::process()
{
 create_map();
 to_cortical(); 
 if(inv)
  to_cartesian();
}


void LPBilinearGpu::create_map(){
 hipMalloc((void**)&xc_d, R*S*sizeof(float));
 hipMalloc((void**)&yc_d, R*S*sizeof(float));
 dim3 dimBlock(BLOCKSZ, BLOCKSZ);
 dim3 dimCGrid(R/dimBlock.x+1, S/dimBlock.y+1);
 
 kernels::createCorticalMapKernel<<<dimCGrid, dimBlock>>>(x0,y0,a,q,p0, xc_d,yc_d,R,S);


if (inv)
 {
  hipMalloc((void**)&e_d, W*H*sizeof(float));
  hipMalloc((void**)&n_d, W*H*sizeof(float));
  dim3 dimRGrid(W/dimBlock.x+1, H/dimBlock.y+1);
  kernels::createRetinalMapKernel<<<dimRGrid, dimBlock>>>(x0,y0,a,q,p0,e_d,n_d,W,H);
 }
}





void LPBilinearGpu::to_cortical(){
 int *cort=new int[R*S];

  dim3 dimBlock(BLOCKSZ, BLOCKSZ);
  dim3 dimGrid(R/dimBlock.x+1, S/dimBlock.y+1);
 
 kernels::interpKernel<<<dimGrid, dimBlock>>>(imgfilter->GetGpuCPnt(), xc_d, yc_d, W, H, R, S,true, imgfilter->GetGpuRPnt());

 hipMemcpy(cort, imgfilter->GetGpuCPnt(), R*S*sizeof(int), hipMemcpyDeviceToHost);

 imgfilter->SetData(R,S,cort);
 delete [] cort;
}

void LPBilinearGpu::to_cartesian(){
 int *ret= new int [W*H];

  dim3 dimBlock(BLOCKSZ, BLOCKSZ);
  dim3 dimGrid(W/dimBlock.x+1, H/dimBlock.y+1);
 
 kernels::interpKernel<<<dimGrid, dimBlock>>>(imgfilter->GetGpuRPnt(), e_d, n_d, R, S, W, H,false, imgfilter->GetGpuCPnt());

 hipMemcpy(ret, imgfilter->GetGpuRPnt(), W*H*sizeof(int), hipMemcpyDeviceToHost);


 imgfilter->SetData(W,H,ret);
 delete [] ret;
}

} // namespace processing
} // namespace rtlp
